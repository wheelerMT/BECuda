#include "hip/hip_runtime.h"
#include <vector>
#include "constants.h"
#include "grid.cuh"

void Grid2D::constructGridParameters()
{
    // K-space grid spacing
    xFourierGridSpacing = PI / (xNumGridPts / 2. * xGridSpacing);
    yFourierGridSpacing = PI / (yNumGridPts / 2. * yGridSpacing);

    // Set length of sides of box
    xLengthOfBox = xNumGridPts * xGridSpacing;
    yLengthOfBox = yNumGridPts * yGridSpacing;
}

void Grid2D::constructGrids()
{
    // Allocate meshgrid arrays
    xMesh = new double[xNumGridPts * yNumGridPts];
    yMesh = new double[xNumGridPts * yNumGridPts];
    xFourierMesh = new double[xNumGridPts * yNumGridPts];
    yFourierMesh = new double[xNumGridPts * yNumGridPts];
    wavenumberMesh = new double[xNumGridPts * yNumGridPts];

    // Construct grids
    for (int i = 0; i < xNumGridPts; ++i)
    {
        for (int j = 0; j < yNumGridPts; ++j)
        {
            xMesh[j + i * yNumGridPts] = (j - xNumGridPts / 2.) * xGridSpacing;
            xFourierMesh[j + i * yNumGridPts] = (j - yNumGridPts / 2.) * xFourierGridSpacing;
            yMesh[j + i * yNumGridPts] = (j - xNumGridPts / 2.) * yGridSpacing;
            yFourierMesh[j + i * yNumGridPts] = (j - yNumGridPts / 2.) * yFourierGridSpacing;
            wavenumberMesh[j + i * yNumGridPts] = std::pow(xFourierMesh[j + i * yNumGridPts], 2)
                                                  + std::pow(yFourierMesh[j + i * yNumGridPts], 2);
        }
    }
}

void Grid2D::fftshift() const
{
    /*
    * Shifts the zero-frequency component to the center
    * of the spectrum.
    */

    std::vector<double> xFourierMeshCopy(xNumGridPts * yNumGridPts);
    std::vector<double> yFourierMeshCopy(xNumGridPts * yNumGridPts);

    for (int i = 0; i < xNumGridPts; ++i)
    {
        for (int j = 0; j < yNumGridPts; ++j)
        {
            xFourierMeshCopy[j + i * xNumGridPts] = xFourierMesh[j + i * xNumGridPts];
            yFourierMeshCopy[j + i * xNumGridPts] = yFourierMesh[j + i * xNumGridPts];
        }
    }

    // Reverse each row
    for (int i = 0; i < xNumGridPts; ++i)
    {
        for (int j = 0; j < xNumGridPts; ++j)
        {
            if (j < xNumGridPts / 2)
            {
                xFourierMesh[j + i * yNumGridPts] = xFourierMeshCopy[xNumGridPts / 2 + j + i * yNumGridPts];
                yFourierMesh[j + i * yNumGridPts] = yFourierMeshCopy[xNumGridPts / 2 + j + i * yNumGridPts];
            } else if (j >= xNumGridPts / 2)
            {
                xFourierMesh[j + i * yNumGridPts] = xFourierMeshCopy[j - xNumGridPts / 2 + i * yNumGridPts];
                yFourierMesh[j + i * yNumGridPts] = yFourierMeshCopy[j - xNumGridPts / 2 + i * yNumGridPts];
            }

        }
    }

    for (int i = 0; i < xNumGridPts; ++i)
    {
        for (int j = 0; j < yNumGridPts; ++j)
        {
            xFourierMeshCopy[j + i * xNumGridPts] = xFourierMesh[j + i * xNumGridPts];
            yFourierMeshCopy[j + i * xNumGridPts] = yFourierMesh[j + i * xNumGridPts];
        }
    }

    // Reverse each column
    for (int i = 0; i < xNumGridPts; ++i)
    {
        for (int j = 0; j < xNumGridPts; ++j)
        {
            if (j < xNumGridPts / 2)
            {
                xFourierMesh[i + j * xNumGridPts] = xFourierMeshCopy[(xNumGridPts / 2 + j) * xNumGridPts + i];
                yFourierMesh[i + j * xNumGridPts] = yFourierMeshCopy[(xNumGridPts / 2 + j) * xNumGridPts + i];
            } else if (j >= xNumGridPts / 2)
            {
                xFourierMesh[i + j * xNumGridPts] = xFourierMeshCopy[(j - xNumGridPts / 2) * xNumGridPts + i];
                yFourierMesh[i + j * xNumGridPts] = yFourierMeshCopy[(j - xNumGridPts / 2) * xNumGridPts + i];
            }
        }
    }

    // Re-update wavenumber, k
    for (int i = 0; i < xNumGridPts; ++i)
    {
        for (int j = 0; j < xNumGridPts; ++j)
        {
            wavenumberMesh[j + i * xNumGridPts] =
                    std::pow(xFourierMesh[j + i * xNumGridPts], 2) + std::pow(yFourierMesh[j + i * xNumGridPts], 2);
        }
    }
}

Grid2D::Grid2D(int xNumGridPts, int yNumGridPts, double xGridSpacing, double yGridSpacing)
        : xNumGridPts{xNumGridPts}, yNumGridPts{yNumGridPts},
          xGridSpacing{xGridSpacing}, yGridSpacing{yGridSpacing}
{
    constructGridParameters();
    constructGrids();
}

Grid2D::Grid2D(const Grid2D &grid)
        : xNumGridPts{grid.xNumGridPts}, yNumGridPts{grid.yNumGridPts},
          xGridSpacing{grid.xGridSpacing}, yGridSpacing{grid.yGridSpacing}
{
    constructGridParameters();
    constructGrids();
}

Grid2D::~Grid2D()
{
    delete[] xMesh;
    delete[] yMesh;
    delete[] xFourierMesh;
    delete[] yFourierMesh;
    delete[] wavenumberMesh;
}
