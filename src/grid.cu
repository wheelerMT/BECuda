#include "hip/hip_runtime.h"
//
// Created by mattw on 27/01/2022.
//

#include <vector>
#include "constants.h"
#include "grid.cuh"

void Grid2D::constructGridParameters()
{
    // K-space grid spacing
    dkx = PI / (nx / 2. * dx);
    dky = PI / (ny / 2. * dy);

    // Set length of sides of box
    lenX = nx * dx;
    lenY = ny * dy;
}

void Grid2D::constructGrids()
{
    // Allocate meshgrid arrays
    X = new double[nx * ny];
    Y = new double[nx * ny];
    Kx = new double[nx * ny];
    Ky = new double[nx * ny];
    K = new double[nx * ny];

    // Construct grids
    for (int i = 0; i < nx; ++i)
    {
        for (int j = 0; j < ny; ++j)
        {
            X[j + i * ny] = (j - nx / 2.) * dx;
            Kx[j + i * ny] = (j - ny / 2.) * dkx;
            Y[j + i * ny] = (j - nx / 2.) * dy;
            Ky[j + i * ny] = (j - ny / 2.) * dky;
            K[j + i * ny] = std::pow(Kx[j + i * ny], 2) + std::pow(Ky[j + i * ny], 2);
        }
    }
}

void Grid2D::fftshift() const
{
    /*
        Shifts the zero-frequency component to the center
        of the spectrum.
    */

    // Make a copy of K-space arrays
    std::vector<double> kxCopy(nx * ny);
    std::vector<double> kyCopy(nx * ny);

    for (int i = 0; i < nx; ++i)
    {
        for (int j = 0; j < ny; ++j)
        {
            kxCopy[j + i * nx] = Kx[j + i * nx];
            kyCopy[j + i * nx] = Ky[j + i * nx];
        }
    }

    // Reverse each row
    for (int i = 0; i < nx; ++i)
    {
        for (int j = 0; j < nx; ++j)
        {
            if (j < nx / 2)
            {
                Kx[j + i * ny] = kxCopy[nx / 2 + j + i * ny];
                Ky[j + i * ny] = kyCopy[nx / 2 + j + i * ny];
            } else if (j >= nx / 2)
            {
                Kx[j + i * ny] = kxCopy[j - nx / 2 + i * ny];
                Ky[j + i * ny] = kyCopy[j - nx / 2 + i * ny];
            }

        }
    }

    // Update array copies
    for (int i = 0; i < nx; ++i)
    {
        for (int j = 0; j < ny; ++j)
        {
            kxCopy[j + i * nx] = Kx[j + i * nx];
            kyCopy[j + i * nx] = Ky[j + i * nx];
        }
    }

    // Reverse each column
    for (int i = 0; i < nx; ++i)
    {
        for (int j = 0; j < nx; ++j)
        {
            if (j < nx / 2)
            {
                Kx[i + j * nx] = kxCopy[(nx / 2 + j) * nx + i];
                Ky[i + j * nx] = kyCopy[(nx / 2 + j) * nx + i];
            } else if (j >= nx / 2)
            {
                Kx[i + j * nx] = kxCopy[(j - nx / 2) * nx + i];
                Ky[i + j * nx] = kyCopy[(j - nx / 2) * nx + i];
            }
        }
    }

    // Re-update wavenumber, k
    for (int i = 0; i < nx; ++i)
    {
        for (int j = 0; j < nx; ++j)
        {
            K[j + i * nx] = std::pow(Kx[j + i * nx], 2) + std::pow(Ky[j + i * nx], 2);
        }
    }
}

Grid2D::Grid2D(unsigned int nx, unsigned int ny, double dx, double dy)
        : nx{nx}, ny{ny}, dx{dx}, dy{dy}
{
    constructGridParameters();
    constructGrids();
}

Grid2D::~Grid2D()
{
    // Delete dynamically allocated arrays
    delete[] X;
    delete[] Y;
    delete[] Kx;
    delete[] Ky;
    delete[] K;
}
