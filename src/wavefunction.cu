#include "wavefunction.cuh"


Wavefunction2D::Wavefunction2D(Grid2D &grid) : grid{grid}
{
    hipMallocManaged(&plusComponent, grid.xNumGridPts * grid.yNumGridPts * sizeof(hipfftComplex));
    hipMallocManaged(&zeroComponent, grid.xNumGridPts * grid.yNumGridPts * sizeof(hipfftComplex));
    hipMallocManaged(&minusComponent, grid.xNumGridPts * grid.yNumGridPts * sizeof(hipfftComplex));
    hipMallocManaged(&plusFourierComponent, grid.xNumGridPts * grid.yNumGridPts * sizeof(hipfftComplex));
    hipMallocManaged(&zeroFourierComponent, grid.xNumGridPts * grid.yNumGridPts * sizeof(hipfftComplex));
    hipMallocManaged(&minusFourierComponent, grid.xNumGridPts * grid.yNumGridPts * sizeof(hipfftComplex));

    trappingPotential = new double[grid.xNumGridPts * grid.yNumGridPts]{};

    generateFFTPlans();
}

void Wavefunction2D::generateFFTPlans()
{
    hipfftPlan2d(&fftPlan, grid.xNumGridPts, grid.yNumGridPts, HIPFFT_C2C);
}

Wavefunction2D::~Wavefunction2D()
{
    hipfftDestroy(fftPlan);

    hipFree(plusComponent);
    hipFree(zeroComponent);
    hipFree(minusComponent);
    hipFree(plusFourierComponent);
    hipFree(zeroFourierComponent);
    hipFree(minusFourierComponent);

    delete[] trappingPotential;

}

void Wavefunction2D::setTrappingPotential(const double *newTrappingPotential) const
{
    for (int i = 0; i < grid.xNumGridPts; ++i)
    {
        for (int j = 0; j < grid.xNumGridPts; ++j)
        {
            trappingPotential[j + i * grid.yNumGridPts] = newTrappingPotential[j + i * grid.yNumGridPts];
        }
    }
}

void Wavefunction2D::setInitialState(const std::string &groundState) const
{
    if (groundState == "polar")
    {
        setPolarInitialState();
    }

    // Can add more ground states as needed
}

void Wavefunction2D::setPolarInitialState() const
{
    for (int i = 0; i < grid.xNumGridPts; ++i)
    {
        for (int j = 0; j < grid.yNumGridPts; ++j)
        {
            plusComponent[j + i * grid.xNumGridPts] = {0., 0.};
            zeroComponent[j + i * grid.xNumGridPts] = {1., 0.};
            minusComponent[j + i * grid.xNumGridPts] = {0., 0.};
        }
    }
}

void Wavefunction2D::addNoiseToComponents(const std::string &components, float mean, float stddev) const
{
    // Construct random generator
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator{seed};
    std::normal_distribution<float> norm_dist{mean, stddev};

    if (components == "outer")
    {
        std::cout << "Adding noise...\n";
        for (int i = 0; i < grid.xNumGridPts; i++)
        {
            for (int j = 0; j < grid.yNumGridPts; j++)
            {
                plusComponent[j + i * grid.xNumGridPts].x += norm_dist(generator);
                plusComponent[j + i * grid.xNumGridPts].y += norm_dist(generator);
                minusComponent[j + i * grid.xNumGridPts].x += norm_dist(generator);
                minusComponent[j + i * grid.xNumGridPts].y += norm_dist(generator);
            }
        }
    }

    // Add other component combinations as needed
}

void Wavefunction2D::fft() const
{
    hipfftExecC2C(fftPlan, plusComponent, plusFourierComponent, HIPFFT_FORWARD);
    hipfftExecC2C(fftPlan, zeroComponent, zeroFourierComponent, HIPFFT_FORWARD);
    hipfftExecC2C(fftPlan, minusComponent, minusFourierComponent, HIPFFT_FORWARD);
    hipDeviceSynchronize();
}

void Wavefunction2D::ifft() const
{
    hipfftExecC2C(fftPlan, plusFourierComponent, plusComponent, HIPFFT_BACKWARD);
    hipfftExecC2C(fftPlan, zeroFourierComponent, zeroComponent, HIPFFT_BACKWARD);
    hipfftExecC2C(fftPlan, minusFourierComponent, minusComponent, HIPFFT_BACKWARD);
    hipDeviceSynchronize();
}