//
// Created by mattw on 07/02/2022.
//

#include "wavefunction.cuh"


Wavefunction2D::Wavefunction2D(Grid2D &grid) : grid{grid}
{
    // Allocate arrays on device
    hipMalloc(&plus, grid.nx * grid.ny * sizeof(hipfftComplex));
    hipMalloc(&zero, grid.nx * grid.ny * sizeof(hipfftComplex));
    hipMalloc(&minus, grid.nx * grid.ny * sizeof(hipfftComplex));
    hipMalloc(&plus_k, grid.nx * grid.ny * sizeof(hipfftComplex));
    hipMalloc(&zero_k, grid.nx * grid.ny * sizeof(hipfftComplex));
    hipMalloc(&minus_k, grid.nx * grid.ny * sizeof(hipfftComplex));

    // Initialise FFT plans
    generateFFTPlans();
}

Wavefunction2D::~Wavefunction2D()
{
    // Free device memory
    hipFree(plus);
    hipFree(zero);
    hipFree(minus);
    hipFree(plus_k);
    hipFree(zero_k);
    hipFree(minus_k);
}

void Wavefunction2D::generateFFTPlans()
{
    // Generate CUDA FFT plans for each component
    hipfftPlan2d(&m_FFTPlan, grid.nx, grid.ny, HIPFFT_C2C);

}

void Wavefunction2D::setInitialState(const std::string &gsPhase)
{
    if (gsPhase == "polar")
    {
        for (int i = 0; i < grid.nx; ++i)
        {
            for (int j = 0; j < grid.ny; ++j)
            {
                plus[j + i * grid.nx] = {0., 0.};
                zero[j + i * grid.nx] = {1., 0.};
                minus[j + i * grid.nx] = {0., 0.};
            }
        }
    }
}
