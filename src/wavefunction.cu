//
// Created by mattw on 07/02/2022.
//

#include "wavefunction.cuh"


Wavefunction2D::Wavefunction2D(Grid2D &grid) : grid{grid}
{
    // Allocate arrays
    plus = new hipfftComplex[grid.nx * grid.ny]{};
    zero = new hipfftComplex[grid.nx * grid.ny]{};
    minus = new hipfftComplex[grid.nx * grid.ny]{};
    plus_k = new hipfftComplex[grid.nx * grid.ny]{};
    zero_k = new hipfftComplex[grid.nx * grid.ny]{};
    minus_k = new hipfftComplex[grid.nx * grid.ny]{};

    // Initialise FFT plans
    generateFFTPlans();
}

Wavefunction2D::~Wavefunction2D()
{
    // Free device memory
    hipFree(plus);
    hipFree(zero);
    hipFree(minus);
    hipFree(plus_k);
    hipFree(zero_k);
    hipFree(minus_k);
}

void Wavefunction2D::generateFFTPlans()
{
    // Generate CUDA FFT plans for each component
    hipfftPlan2d(&fftPlan, grid.nx, grid.ny, HIPFFT_C2C);

}

void Wavefunction2D::setInitialState(const std::string &groundState) const
{
    if (groundState == "polar")
    {
        setPolarInitialState();
    }

    // Can add more ground states as needed
}

void Wavefunction2D::setPolarInitialState() const
{
    for (int i = 0; i < grid.nx; ++i)
    {
        for (int j = 0; j < grid.ny; ++j)
        {
            plus[j + i * grid.nx] = {0., 0.};
            zero[j + i * grid.nx] = {1., 0.};
            minus[j + i * grid.nx] = {0., 0.};
        }
    }
}

void Wavefunction2D::add_noise(const std::string &components, float mean, float stddev) const
{
    // Construct random generator
    unsigned seed1 = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator{seed1};
    std::normal_distribution<float> norm_dist{mean, stddev};

    if (components == "outer")
    {
        std::cout << "Adding noise...\n";

        // Add noise to outer components
        for (int i = 0; i < grid.nx; i++)
        {
            for (int j = 0; j < grid.ny; j++)
            {
                plus[j + i * grid.nx].x += norm_dist(generator);
                plus[j + i * grid.nx].y += norm_dist(generator);
                minus[j + i * grid.nx].x += norm_dist(generator);
                minus[j + i * grid.nx].y += norm_dist(generator);
            }
        }
    }

    // Add other component combinations as needed
}
