//
// Created by mattw on 07/02/2022.
//

#include "wavefunction.cuh"


Wavefunction2D::Wavefunction2D(Grid2D &grid) : grid{grid}
{
    // Allocate arrays
    plusComponent = new hipfftComplex[grid.nx * grid.ny]{};
    zeroComponent = new hipfftComplex[grid.nx * grid.ny]{};
    minusComponent = new hipfftComplex[grid.nx * grid.ny]{};
    plusFourierComponent = new hipfftComplex[grid.nx * grid.ny]{};
    zeroFourierComponent = new hipfftComplex[grid.nx * grid.ny]{};
    minusFourierComponent = new hipfftComplex[grid.nx * grid.ny]{};
}

Wavefunction2D::~Wavefunction2D()
{
    // Free device memory
    hipFree(plusComponent);
    hipFree(zeroComponent);
    hipFree(minusComponent);
    hipFree(plusFourierComponent);
    hipFree(zeroFourierComponent);
    hipFree(minusFourierComponent);
}

void Wavefunction2D::generateFFTPlans()
{
    // Generate CUDA FFT plans for each component
    hipfftPlan2d(&fftPlan, grid.nx, grid.ny, HIPFFT_C2C);

}

void Wavefunction2D::setInitialState(const std::string &groundState) const
{
    if (groundState == "polar")
    {
        setPolarInitialState();
    }

    // Can add more ground states as needed
}

void Wavefunction2D::setPolarInitialState() const
{
    for (int i = 0; i < grid.nx; ++i)
    {
        for (int j = 0; j < grid.ny; ++j)
        {
            plusComponent[j + i * grid.nx] = {0., 0.};
            zeroComponent[j + i * grid.nx] = {1., 0.};
            minusComponent[j + i * grid.nx] = {0., 0.};
        }
    }
}

void Wavefunction2D::addNoise(const std::string &components, float mean, float stddev) const
{
    // Construct random generator
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator{seed};
    std::normal_distribution<float> norm_dist{mean, stddev};

    if (components == "outer")
    {
        std::cout << "Adding noise...\n";

        // Add noise to outer components
        for (int i = 0; i < grid.nx; i++)
        {
            for (int j = 0; j < grid.ny; j++)
            {
                plusComponent[j + i * grid.nx].x += norm_dist(generator);
                plusComponent[j + i * grid.nx].y += norm_dist(generator);
                minusComponent[j + i * grid.nx].x += norm_dist(generator);
                minusComponent[j + i * grid.nx].y += norm_dist(generator);
            }
        }
    }

    // Add other component combinations as needed
}
