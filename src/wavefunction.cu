//
// Created by mattw on 07/02/2022.
//

#include "wavefunction.cuh"


Wavefunction2D::Wavefunction2D(Grid2D &grid) : grid{grid}
{
    // Allocate arrays
    plus = new hipfftComplex[grid.nx * grid.ny];
    zero = new hipfftComplex[grid.nx * grid.ny];
    minus = new hipfftComplex[grid.nx * grid.ny];
    plus_k = new hipfftComplex[grid.nx * grid.ny];
    zero_k = new hipfftComplex[grid.nx * grid.ny];
    minus_k = new hipfftComplex[grid.nx * grid.ny];

    // Initialise FFT plans
    generateFFTPlans();
}

Wavefunction2D::~Wavefunction2D()
{
    // This needs to call appropriate functions
    // to de-allocate arrays on device memory
}

void Wavefunction2D::generateFFTPlans()
{
    // Generate CUDA FFT plans for each component check for errors
    if (hipfftPlan2d(&m_planPlus, grid.nx, grid.ny, HIPFFT_C2C) != HIPFFT_SUCCESS)
    {
        std::cerr << "CUFFT error: Plan creation failed...\n";
        return;
    };
    if (hipfftPlan2d(&m_planZero, grid.nx, grid.ny, HIPFFT_C2C) != HIPFFT_SUCCESS)
    {
        std::cerr << "CUFFT error: Plan creation failed...\n";
        return;
    };
    if (hipfftPlan2d(&m_planMinus, grid.nx, grid.ny, HIPFFT_C2C) != HIPFFT_SUCCESS)
    {
        std::cerr << "CUFFT error: Plan creation failed...\n";
        return;
    };
}

void Wavefunction2D::setInitialState(const std::string &gsPhase)
{
    if (gsPhase == "polar")
    {
        for (int i = 0; i < grid.nx; ++i)
        {
            for (int j = 0; j < grid.ny; ++j)
            {
                plus[j + i * grid.nx] = {0., 0.};
                zero[j + i * grid.nx] = {1., 0.};
                minus[j + i * grid.nx] = {0., 0.};
            }
        }
    }
}
