#include "wavefunction.cuh"


Wavefunction2D::Wavefunction2D(Grid2D &grid) : grid{grid}
{
    plusComponent = new hipfftComplex[grid.xNumGridPts * grid.yNumGridPts]{};
    zeroComponent = new hipfftComplex[grid.xNumGridPts * grid.yNumGridPts]{};
    minusComponent = new hipfftComplex[grid.xNumGridPts * grid.yNumGridPts]{};
    plusFourierComponent = new hipfftComplex[grid.xNumGridPts * grid.yNumGridPts]{};
    zeroFourierComponent = new hipfftComplex[grid.xNumGridPts * grid.yNumGridPts]{};
    minusFourierComponent = new hipfftComplex[grid.xNumGridPts * grid.yNumGridPts]{};
}

Wavefunction2D::~Wavefunction2D()
{
    hipFree(plusComponent);
    hipFree(zeroComponent);
    hipFree(minusComponent);
    hipFree(plusFourierComponent);
    hipFree(zeroFourierComponent);
    hipFree(minusFourierComponent);
}

void Wavefunction2D::generateFFTPlans()
{
    hipfftPlan2d(&fftPlan, grid.xNumGridPts, grid.yNumGridPts, HIPFFT_C2C);
}

void Wavefunction2D::setInitialState(const std::string &groundState) const
{
    if (groundState == "polar")
    {
        setPolarInitialState();
    }

    // Can add more ground states as needed
}

void Wavefunction2D::setPolarInitialState() const
{
    for (int i = 0; i < grid.xNumGridPts; ++i)
    {
        for (int j = 0; j < grid.yNumGridPts; ++j)
        {
            plusComponent[j + i * grid.xNumGridPts] = {0., 0.};
            zeroComponent[j + i * grid.xNumGridPts] = {1., 0.};
            minusComponent[j + i * grid.xNumGridPts] = {0., 0.};
        }
    }
}

void Wavefunction2D::addNoise(const std::string &components, float mean, float stddev) const
{
    // Construct random generator
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator{seed};
    std::normal_distribution<float> norm_dist{mean, stddev};

    if (components == "outer")
    {
        std::cout << "Adding noise...\n";
        for (int i = 0; i < grid.xNumGridPts; i++)
        {
            for (int j = 0; j < grid.yNumGridPts; j++)
            {
                plusComponent[j + i * grid.xNumGridPts].x += norm_dist(generator);
                plusComponent[j + i * grid.xNumGridPts].y += norm_dist(generator);
                minusComponent[j + i * grid.xNumGridPts].x += norm_dist(generator);
                minusComponent[j + i * grid.xNumGridPts].y += norm_dist(generator);
            }
        }
    }

    // Add other component combinations as needed
}
