#include "wavefunction.cuh"


Wavefunction2D::Wavefunction2D(Grid2D &grid) : grid{grid}
{
    h_plusComponent = new hipfftComplex[grid.xNumGridPts * grid.yNumGridPts]{};
    h_zeroComponent = new hipfftComplex[grid.xNumGridPts * grid.yNumGridPts]{};
    h_minusComponent = new hipfftComplex[grid.xNumGridPts * grid.yNumGridPts]{};

    trappingPotential = new double[grid.xNumGridPts * grid.yNumGridPts] {};
}

Wavefunction2D::~Wavefunction2D()
{
    delete[] h_plusComponent;
    delete[] h_zeroComponent;
    delete[] h_minusComponent;

    hipFree(plusComponent);
    hipFree(zeroComponent);
    hipFree(minusComponent);
    hipFree(plusFourierComponent);
    hipFree(zeroFourierComponent);
    hipFree(minusFourierComponent);
}

void Wavefunction2D::setTrappingPotential(const double *newTrappingPotential) const
{
    for (int i = 0; i < grid.xNumGridPts; ++i)
    {
        for (int j = 0; j < grid.xNumGridPts; ++j)
        {
            trappingPotential[j + i * grid.yNumGridPts] = newTrappingPotential[j + i * grid.yNumGridPts];
        }
    }
}

void Wavefunction2D::generateFFTPlans()
{
    hipfftPlan2d(&fftPlan, grid.xNumGridPts, grid.yNumGridPts, HIPFFT_C2C);
}

void Wavefunction2D::setInitialState(const std::string &groundState) const
{
    if (groundState == "polar")
    {
        setPolarInitialState();
    }

    // Can add more ground states as needed
}

void Wavefunction2D::setPolarInitialState() const
{
    for (int i = 0; i < grid.xNumGridPts; ++i)
    {
        for (int j = 0; j < grid.yNumGridPts; ++j)
        {
            h_plusComponent[j + i * grid.xNumGridPts] = {0., 0.};
            h_zeroComponent[j + i * grid.xNumGridPts] = {1., 0.};
            h_minusComponent[j + i * grid.xNumGridPts] = {0., 0.};
        }
    }
}

void Wavefunction2D::addNoiseToComponents(const std::string &components, float mean, float stddev) const
{
    // Construct random generator
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator{seed};
    std::normal_distribution<float> norm_dist{mean, stddev};

    if (components == "outer")
    {
        std::cout << "Adding noise...\n";
        for (int i = 0; i < grid.xNumGridPts; i++)
        {
            for (int j = 0; j < grid.yNumGridPts; j++)
            {
                h_plusComponent[j + i * grid.xNumGridPts].x += norm_dist(generator);
                h_plusComponent[j + i * grid.xNumGridPts].y += norm_dist(generator);
                h_minusComponent[j + i * grid.xNumGridPts].x += norm_dist(generator);
                h_minusComponent[j + i * grid.xNumGridPts].y += norm_dist(generator);
            }
        }
    }

    // Add other component combinations as needed
}
